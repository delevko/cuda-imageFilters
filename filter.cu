
#include <hip/hip_runtime.h>
#include<cstdio>

struct RGB {
    unsigned char r, g, b, a;
};

extern "C" {
	__global__
	void BlackWhite(RGB *image, int width, int height) {
		int thidx = blockIdx.x*blockDim.x + threadIdx.x;

		if (thidx >= width*height) { 
			return;
		}

		unsigned char tmp = 0.299*image[thidx].r + 
		    0.587*image[thidx].g + 0.114*image[thidx].b;
		image[thidx].r = image[thidx].g = image[thidx].b = tmp;
	}

	__global__
	void Negative(RGB *oldImage, RGB *newImage, int width, int height)
    {
		int thidx = blockIdx.x*blockDim.x + threadIdx.x;
        
        if(thidx >= width*height) {
            return;
        }

        RGB tmp = oldImage[thidx];
        tmp.r = (unsigned char) 255 - tmp.r;
        tmp.g = (unsigned char) 255 - tmp.g;
        tmp.b = (unsigned char) 255 - tmp.b;

        newImage[thidx] = tmp;
    }

	__global__
	void Normalization(RGB *image, int width, int height) {
		const int thidx = blockIdx.x*blockDim.x + threadIdx.x;
		if (thidx >= width*height) { 
			return;
		}

		int tmp = image[thidx].r + image[thidx].g + image[thidx].b;
		image[thidx].r = (unsigned char) (image[thidx].r*255.0 / tmp);
		image[thidx].g = (unsigned char) (image[thidx].g*255.0 / tmp);
		image[thidx].b = (unsigned char) (image[thidx].b*255.0 / tmp);
	}
}
